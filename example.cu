#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#define N 10

__global__ void kernel(int* d_vec, int n) {

    int tid = threadIdx.x;

    if(threadIdx.x < n) {
        int i = d_vec[tid];
        d_vec[tid] = i > 5 ? -i : i;
    }
}

int main(int argc, char** argv){

    hipMemcpy(d_msg, msg_in, N*sizeof(int), hipMemcpyHostToDevice);
    kernel<<<1,100>>>(d_msg, N);
    hipMemcpy(msg_out, d_msg, N*sizeof(int), hipMemcpyDeviceToHost);
    
}
