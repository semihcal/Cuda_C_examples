
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10 

int main(int argc, char** argv){
    int vec_in[N] ={6,1,7,3,2,9,10,5,4,8};

    int vec_out[N];

    int* d_vec;
    

    hipMalloc(&d_vec, N*sizeof(int));

    hipMemcpy(d_vec, vec_in, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vec_out, d_vec, N*sizeof(int), hipMemcpyDeviceToHost);

    printf("vec_out[3]= %d \n",vec_out[3]);

    

    return 0;
}
